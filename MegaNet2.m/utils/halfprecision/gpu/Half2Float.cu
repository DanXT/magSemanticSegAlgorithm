#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_fp16.h>

/*
 * Device code
 */
void __global__ float2half(__half const * const A, float * const B, int const N)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
    int const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        B[i] =  __half2float(A[i]);
    }
}
/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    mxGPUArray const *A;
    mxGPUArray *B;
    __half const *d_A;
    float *d_B = NULL;
    int N;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";
	
	/* Choose a reasonably sized number of threads for the block. */
    int const threadsPerBlock = 256;
    int blocksPerGrid;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
    if ((nrhs!=1) || !(mxIsGPUArray(prhs[0]))) {
        mexErrMsgIdAndTxt(errId, "input is not a GPU array");
    }
    

    A = mxGPUCreateFromMxArray(prhs[0]);

    /*
     * Verify that A really is a float array before extracting the pointer.
     */
    if (mxGPUGetClassID(A) != mxUINT16_CLASS) {
        mexErrMsgIdAndTxt(errId, "A is not a single class");
    }

    /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_A = (__half const *)(mxGPUGetDataReadOnly(A));

    /* Create a GPUArray to hold the result and get its underlying pointer. */
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxSINGLE_CLASS,
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float*)mxGPUGetData(B);

    /*
     * Call the kernel using the CUDA runtime API. We are using a 1-d grid here,
     * and it would be possible for the number of elements to be too large for
     * the grid. For this example we are not guarding against this possibility.
     */
    N = (int)(mxGPUGetNumberOfElements(A));
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    float2half<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(B);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(B);
}
